#include "hip/hip_runtime.h"
#include <stdint.h>
#include <stdlib.h>
#include <string.h>

#include <cutil.h>
#include "util.h"
#include "ref_2dhisto.h"

#define T 12

__global__ void opt_2dhistoKernel(uint32_t*, size_t, size_t, uint32_t*);
__global__ void opt_32to8Kernel(uint32_t*, uint8_t*, size_t);

void opt_2dhisto(uint32_t* input, size_t height, size_t width, uint8_t* bins, uint32_t* g_bins)
{
    /* This function should only contain a call to the GPU 
       histogramming kernel. Any memory allocations and
       transfers must be done outside this function */

	// working
	dim3 block(16, 16);
	dim3 grid(((INPUT_WIDTH + 128) & 0xFFFFFF80) / 16, INPUT_HEIGHT / 16);
	opt_2dhistoKernel<<<grid, block>>>(input, height, width, g_bins);

	hipDeviceSynchronize();

	opt_32to8Kernel<<<HISTO_HEIGHT * HISTO_WIDTH / 512, 512>>>(g_bins, bins, 1024);

	hipDeviceSynchronize();
}

/* Include below the implementation of any other functions you need */
/* kernel verson 1: basic */
__global__ void opt_2dhistoKernel(uint32_t *input, size_t height, size_t width, uint32_t* bins){

	/*
	// working
	int col = blockDim.x * blockIdx.x + threadIdx.x;
	int row = blockDim.y * blockIdx.y + threadIdx.y;

	if (row == 0 && col < 1024) {
		bins[col] = 0;
	}

	__syncthreads();
	if (row < height && col < width) {
		atomicAdd(&bins[input[col + row * ((INPUT_WIDTH + 128) & 0xFFFFFF80)]], 1);
	}
	*/

	/*	
	// working
	__shared__ int temp[1024];
	temp[threadIdx.x] = 0;
	__syncthreads();

	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int offset = blockDim.x * gridDim.x;
	int size = HISTO_HEIGHT * HISTO_WIDTH;

	while (i < size) {
		atomicAdd(&temp[input[i]], 1);
		i += offset;
	} 
	__syncthreads();

	atomicAdd(&(bins[threadIdx.x]), temp[threadIdx.x]);	
	*/

	/*
	// working but slower
	__shared__ int temp[1024];
	temp[threadIdx.x] = 0;
	temp[threadIdx.x + 256] = 0;
	temp[threadIdx.x + 512] = 0;
	temp[threadIdx.x + 768] = 0;
	__syncthreads();

	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int offset = blockDim.x * gridDim.x;
	int size = HISTO_HEIGHT * HISTO_WIDTH;

	while (i < size) {
		atomicAdd(&temp[input[i]], 1);
		i += offset;
	} 
	__syncthreads();

	atomicAdd(&(bins[threadIdx.x]), temp[threadIdx.x]);
	atomicAdd(&(bins[threadIdx.x + 256]), temp[threadIdx.x + 256]);
	atomicAdd(&(bins[threadIdx.x + 512]), temp[threadIdx.x + 512]);
	atomicAdd(&(bins[threadIdx.x + 768]), temp[threadIdx.x + 768]);
	*/

	int col = blockDim.x * blockIdx.x + threadIdx.x;
	int row = blockDim.y * blockIdx.y + threadIdx.y;
	int mask = (INPUT_WIDTH + 128) & 0xFFFFFF80;

	if (row == 0 && col < 1024) {
		bins[col] = 0;
	}

	int index;
	__syncthreads();
	if (row < height && col < width) {
		index = input[col + row * mask];
		if (bins[index] < 255)
			atomicAdd(&bins[index], 1);
	}
		
}

__global__ void opt_32to8Kernel(uint32_t *input, uint8_t* output, size_t length){
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	
	output[idx] = (uint8_t)((input[idx] < UINT8_MAX) * input[idx]) + (input[idx] >= UINT8_MAX) * UINT8_MAX;

	__syncthreads();
}

void* AllocateDevice(size_t size){
	void* ret;
	hipMalloc(&ret, size);
	return ret;
}

void CopyToDevice(void* D_device, void* D_host, size_t size){
	hipMemcpy(D_device, D_host, size, 
					hipMemcpyHostToDevice);
}

void CopyFromDevice(void* D_host, void* D_device, size_t size){
	hipMemcpy(D_host, D_device, size, 
					hipMemcpyDeviceToHost);
}

void FreeDevice(void* D_device){
	hipFree(D_device);
}
