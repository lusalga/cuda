#include "hip/hip_runtime.h"
#include <stdint.h>
#include <stdlib.h>
#include <string.h>
#include <stdio.h>

#include <cutil.h>
#include "util.h"
#include "ref_2dhisto.h"


__global__ void HistKernel(uint32_t *deviceImage, uint32_t *deviceBins32, size_t height, size_t width) {

	size_t globalTid = blockIdx.x * blockDim.x + threadIdx.x;
	size_t numThreads = blockDim.x * gridDim.x;


	// init histogram for each block
	__shared__ uint32_t partialHist[HISTO_WIDTH + 1];

	partialHist[threadIdx.x] = 0;
	__syncthreads();

	for (size_t j = globalTid; j < height * width; j += numThreads) {
		uint32_t value = deviceImage[j];

		if (partialHist[value] < UINT8_MAX) {
			atomicAdd(&partialHist[value], 1);
		}
	}
	__syncthreads();

	// sum partials
	if (deviceBins32[threadIdx.x] < UINT8_MAX) {
		atomicAdd(&deviceBins32[threadIdx.x], partialHist[threadIdx.x]);
	}
}

__global__ void HistKernel32to8(uint32_t *deviceBins32, uint8_t *deviceBins) {
	// convert int32 to int8; overloaded __nv_min function
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	deviceBins[index] = (uint8_t) min(deviceBins32[index], UINT8_MAX);
}


void opt_2dhisto(uint32_t *deviceImage, uint32_t *deviceBins32, uint8_t *deviceBins, size_t height, size_t width) {
	hipMemset(deviceBins32, 0, HISTO_HEIGHT * HISTO_WIDTH * sizeof(uint32_t)); //zeros

	// Occupancy calculator: 8 multiprocessors * 2 blocks

	HistKernel <<<16, HISTO_WIDTH>>> (deviceImage, deviceBins32, height, width);
	HistKernel32to8 <<<HISTO_HEIGHT, HISTO_WIDTH>>> (deviceBins32, deviceBins);
	hipDeviceSynchronize();

}

uint32_t *AllocateDeviceImage(size_t height, size_t width) {
	uint32_t *deviceImage;
	int size = height * width * sizeof(uint32_t);

	hipMalloc((void**)&deviceImage, size);
	return deviceImage;
}

uint8_t *AllocateDeviceBins(size_t height, size_t width) {
	uint8_t *deviceBins;
    int size = height * width * sizeof(uint8_t);

    hipMalloc((void**)&deviceBins, size);
    return deviceBins;
}

void FreeDeviceImage(uint32_t *deviceImage) {
	hipFree(deviceImage);
	deviceImage = NULL;
}

void FreeDeviceBins(uint8_t *deviceBins) {
	hipFree(deviceBins);
	deviceBins = NULL;
}

void ToDeviceImage(uint32_t *deviceImage, uint32_t *input[],  size_t height, size_t width) {
	int size = width * sizeof(uint32_t);
	for (int i = 0; i < height; i++) {
		hipMemcpy(deviceImage + i * width, input[i], size, hipMemcpyHostToDevice);
	}
}

void ToDeviceBins(uint8_t *deviceBins, uint8_t *hostBins, size_t height, size_t width) {
	int size = height * width * sizeof(uint8_t);
    hipMemcpy(deviceBins, hostBins, size, hipMemcpyHostToDevice);
}

void FromDeviceImage(uint32_t *hostImage, uint32_t *deviceImage, size_t height, size_t width) {
	int size = height * width * sizeof(uint32_t);
    hipMemcpy(hostImage, deviceImage, size, hipMemcpyDeviceToHost);
}

void FromDeviceBins(uint8_t *hostBins, uint8_t *deviceBins, size_t height, size_t width) {
	int size = height * width * sizeof(uint8_t);
    hipMemcpy(hostBins, deviceBins, size, hipMemcpyDeviceToHost);
}
