#include "hip/hip_runtime.h"
#include <stdint.h>
#include <stdlib.h>
#include <string.h>
#include <stdio.h>

#include <cutil.h>
#include "util.h"
#include "ref_2dhisto.h"


__global__ void HistKernel(uint32_t *deviceImage, uint32_t *deviceBins32, size_t height, size_t width) {
	size_t globalTid = blockIdx.x * blockDim.x + threadIdx.x;
	size_t numThreads = (height * width)/HISTO_WIDTH;


	for (size_t j = threadIdx.x * numThreads; j < numThreads * (threadIdx.x + 1); j++){
		uint32_t value = deviceImage[j];

		if(deviceBins32[value] < UINT8_MAX) {
			atomicAdd(&deviceBins32[value], 1);
		}
	}
}

__global__ void HistKernel32to8(uint32_t *deviceBins32, uint8_t *deviceBins, size_t height, size_t width) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	deviceBins[index] = (deviceBins32[index] < UINT8_MAX) ? (uint8_t) deviceBins32[index] : (uint8_t) UINT8_MAX;
}

void opt_2dhisto(uint32_t *deviceImage, uint32_t *deviceBins32, uint8_t *deviceBins, size_t height, size_t width) {
	hipMemset(deviceBins32, 0, HISTO_HEIGHT * HISTO_WIDTH * sizeof(uint32_t));
	HistKernel <<<1, HISTO_WIDTH>>> (deviceImage, deviceBins32, height, width);
	HistKernel32to8 <<<HISTO_HEIGHT, HISTO_WIDTH>>> (deviceBins32, deviceBins, height, width);
	hipDeviceSynchronize();
}



uint32_t *AllocateDeviceImage(size_t height, size_t width) {
	uint32_t *deviceImage;
	int size = height * width * sizeof(uint32_t);

	hipMalloc((void**)&deviceImage, size);
	return deviceImage;
}

uint8_t *AllocateDeviceBins(size_t height, size_t width) {
	uint8_t *deviceBins;
    int size = height * width * sizeof(uint8_t);

    hipMalloc((void**)&deviceBins, size);
    return deviceBins;
}

void FreeDeviceImage(uint32_t *deviceImage) {
	hipFree(deviceImage);
	deviceImage = NULL;
}

void FreeDeviceBins(uint8_t *deviceBins) {
	hipFree(deviceBins);
	deviceBins = NULL;
}

void ToDeviceImage(uint32_t *deviceImage, uint32_t *input[],  size_t height, size_t width) {
	int size = width * sizeof(uint32_t);
	for (int i = 0; i < height; i++) {
		hipMemcpy(deviceImage + i * width, input[i], size, hipMemcpyHostToDevice);
	}
}

void ToDeviceBins(uint8_t *deviceBins, uint8_t *hostBins, size_t height, size_t width) {
	int size = height * width * sizeof(uint8_t);
    hipMemcpy(deviceBins, hostBins, size, hipMemcpyHostToDevice);
}

void FromDeviceImage(uint32_t *hostImage, uint32_t *deviceImage, size_t height, size_t width) {
	int size = height * width * sizeof(uint32_t);
    hipMemcpy(hostImage, deviceImage, size, hipMemcpyDeviceToHost);
}

void FromDeviceBins(uint8_t *hostBins, uint8_t *deviceBins, size_t height, size_t width) {
	int size = height * width * sizeof(uint8_t);
    hipMemcpy(hostBins, deviceBins, size, hipMemcpyDeviceToHost);
}

