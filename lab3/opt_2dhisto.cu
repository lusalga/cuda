#include "hip/hip_runtime.h"
#include <stdint.h>
#include <stdlib.h>
#include <string.h>

#include <cutil.h>
#include "util.h"
#include "ref_2dhisto.h"


__global__ void HistKernel(uint32_t *deviceImage, uint8_t *deviceBins, size_t height, size_t width) {

	for(size_t i = 0; i < height; i++) {
		for(size_t j = 0; j < width; j++) {
			const uint32_t value = deviceImage[i * width + j];

			if (deviceBins[value] < UINT8_MAX) {
				deviceBins[value]++;
			}
		}
	}
}

void opt_2dhisto(uint32_t *deviceImage, uint8_t *deviceBins, size_t height, size_t width) {

	dim3 dimGrid(1, 1, 1);
	dim3 dimBlock(1, 1, 1);

	HistKernel <<<dimGrid, dimBlock>>> (deviceImage, deviceBins, height, width);
	hipDeviceSynchronize();

}

uint32_t *AllocateDeviceImage(size_t height, size_t width) {
	uint32_t *deviceImage;
	int size = height * width * sizeof(uint32_t);

	hipMalloc((void**)&deviceImage, size);
	return deviceImage;
}

uint8_t *AllocateDeviceBins(size_t height, size_t width) {
	uint8_t *deviceBins;
    int size = height * width * sizeof(uint8_t);

    hipMalloc((void**)&deviceBins, size);
    return deviceBins;
}

void FreeDeviceImage(uint32_t *deviceImage) {
	hipFree(deviceImage);
	deviceImage = NULL;
}

void FreeDeviceBins(uint8_t *deviceBins) {
	hipFree(deviceBins);
	deviceBins = NULL;
}

void ToDeviceImage(uint32_t *deviceImage, uint32_t *input[],  size_t height, size_t width) {
	int size = width * sizeof(uint32_t);
	for (int i = 0; i < height; i++) {
		hipMemcpy(deviceImage + i * width, input[i], size, hipMemcpyHostToDevice);
	}
}

void ToDeviceBins(uint8_t *deviceBins, uint8_t *hostBins, size_t height, size_t width) {
	int size = height * width * sizeof(uint8_t);
    hipMemcpy(deviceBins, hostBins, size, hipMemcpyHostToDevice);
}

void FromDeviceImage(uint32_t *hostImage, uint32_t *deviceImage, size_t height, size_t width) {
	int size = height * width * sizeof(uint32_t);
    hipMemcpy(hostImage, deviceImage, size, hipMemcpyDeviceToHost);
}

void FromDeviceBins(uint8_t *hostBins, uint8_t *deviceBins, size_t height, size_t width) {
	int size = height * width * sizeof(uint8_t);
    hipMemcpy(hostBins, deviceBins, size, hipMemcpyDeviceToHost);
}
