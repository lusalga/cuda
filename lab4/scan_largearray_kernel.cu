#include "hip/hip_runtime.h"
#ifndef _PRESCAN_CU_
#define _PRESCAN_CU_

// includes, kernels
#include <assert.h>


#define NUM_BANKS 32
#define LOG_NUM_BANKS 5
// Lab4: You can use any other block size you wish.
#define BLOCK_SIZE 1024
#define OFFSET(n) ((n) >> LOG_NUM_BANKS)
// Lab4: Host Helper Functions (allocate your own data structure...)

int nextPowerOf2(float input) {
	return exp2(ceil(log2(input - 1)));
}

bool isPowerOf2(int input) {
	return (input & (input - 1)) == 0;
}

float *setBlockSums(int size) {
	float* blockSums = NULL;

	CUDA_SAFE_CALL( hipMalloc( (void**) &blockSums,  sizeof( float) * size) );
	CUDA_SAFE_CALL( hipMemset( blockSums, 0, sizeof( float) * size) );

	return blockSums;
}

// Lab4: Device Functions



// Lab4: Kernel Functions
__global__ void singleKernel(float *outArray, float *inArray, int numElements) {
	__shared__ float sharedArray[BLOCK_SIZE + OFFSET(BLOCK_SIZE)];


	sharedArray[threadIdx.x + OFFSET(threadIdx.x)] = (threadIdx.x < numElements) ? inArray[threadIdx.x] : 0;
	__syncthreads();

	for(size_t i = 1; i < BLOCK_SIZE; i <<= 1) {
		size_t index = 2 * i * ( threadIdx.x  + 1) - 1;

		if(index < BLOCK_SIZE) {
			sharedArray[index + OFFSET(index)] += sharedArray[index - i  + OFFSET(index - i)];
		}
		__syncthreads();
	}

	if(threadIdx.x == 0) {sharedArray[BLOCK_SIZE - 1 + OFFSET(BLOCK_SIZE - 1)] = 0;}

	for(size_t i = BLOCK_SIZE >> 1; i > 0; i >>= 1) {
		size_t index = 2 * i * ( threadIdx.x  + 1) - 1;

		if(index < BLOCK_SIZE) {
			float temp = sharedArray[index - i + + OFFSET(index - i)];
			sharedArray[index - i + OFFSET(index - i)] = sharedArray[index + OFFSET(index)];
			sharedArray[index + OFFSET(index)] += temp;
		}
		__syncthreads();
	}

	outArray[threadIdx.x] = sharedArray[threadIdx.x + OFFSET(threadIdx.x)];
}


__global__ void upKernel(float *outArray, float *inArray, float *blockSums, int numElements) {
	size_t globalThread = blockIdx.x * blockDim.x + threadIdx.x;

	__shared__ float sharedArray[BLOCK_SIZE + OFFSET(BLOCK_SIZE)];
	sharedArray[threadIdx.x + OFFSET(threadIdx.x)] = (globalThread < numElements) ? inArray[globalThread] : 0;
	__syncthreads();

	for(size_t i = 1; i < BLOCK_SIZE; i <<= 1) {
		size_t index = 2 * i * ( threadIdx.x  + 1) - 1;

		if(index < BLOCK_SIZE) {
			sharedArray[index + OFFSET(index)] += sharedArray[index - i + OFFSET(index - i)];
		}
		__syncthreads();
	}

	if(threadIdx.x == 0) {
		blockSums[blockIdx.x] = sharedArray[BLOCK_SIZE - 1  + OFFSET(BLOCK_SIZE - 1)];
		sharedArray[BLOCK_SIZE - 1 +  + OFFSET(BLOCK_SIZE - 1)] = 0;
	}

	for(size_t i = BLOCK_SIZE >> 1; i > 0; i >>= 1) {
		size_t index = 2 * i * ( threadIdx.x  + 1) - 1;

		if(index < BLOCK_SIZE) {
			float temp = sharedArray[index - i + OFFSET(index - i)];
			sharedArray[index - i + OFFSET(index - i)] = sharedArray[index + OFFSET(index)];
			sharedArray[index + OFFSET(index)] += temp;
		}
		__syncthreads();
	}

	outArray[globalThread] = sharedArray[threadIdx.x  + OFFSET(threadIdx.x)];
}

__global__ void addKernel(float *outArray, float *blockSums) {
	size_t globalThread = blockIdx.x * blockDim.x + threadIdx.x;
	outArray[globalThread] += blockSums[blockIdx.x];
}


// **===-------- Lab4: Modify the body of this function -----------===**
// You may need to make multiple kernel calls, make your own kernel
// function in this file, and then call them from here.
void prescanArray(float *outArray, float *inArray, float *blockSums, float *blockSumsSums, int numElements)
{

	if(numElements <= BLOCK_SIZE) {
		singleKernel <<<1, BLOCK_SIZE>>> (outArray, inArray, numElements);
	} else if (numElements <= BLOCK_SIZE * BLOCK_SIZE) {
		size_t numBlocks = (numElements - 1) / BLOCK_SIZE + 1;

		upKernel<<<numBlocks, BLOCK_SIZE>>> (outArray, inArray, blockSums, numElements);
		singleKernel<<<1, BLOCK_SIZE>>> (blockSums, blockSums, numBlocks);
		addKernel<<<numBlocks, BLOCK_SIZE>>> (outArray, blockSums);

	} else {
		size_t numBlocks = (numElements - 1)/ BLOCK_SIZE + 1;
		size_t numBlockSums = (numBlocks - 1) / BLOCK_SIZE + 1;

		upKernel<<<numBlocks, BLOCK_SIZE>>> (outArray, inArray, blockSums, numElements);
		upKernel<<<numBlockSums, BLOCK_SIZE>>> (blockSums, blockSums, blockSumsSums, numBlocks);

		singleKernel<<<1, BLOCK_SIZE>>> (blockSumsSums, blockSumsSums, numBlocks);

		addKernel<<<numBlockSums, BLOCK_SIZE>>> (blockSums, blockSumsSums);
		addKernel<<<numBlocks, BLOCK_SIZE>>> (outArray, blockSums);
	}

}
// **===-----------------------------------------------------------===**


#endif // _PRESCAN_CU_
