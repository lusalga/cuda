#include "hip/hip_runtime.h"
#ifndef _PRESCAN_CU_
#define _PRESCAN_CU_

// includes, kernels
#include <assert.h>


#define NUM_BANKS 32
#define LOG_NUM_BANKS 5
// Lab4: You can use any other block size you wish.
#define BLOCK_SIZE 1024

// Lab4: Host Helper Functions (allocate your own data structure...)

int nearestPowerOf2(int input) {
	float count = 0;
	input -= 1;
	if (input == 0) {return 0;}
	while (input > 0) {
		input >>= 1;
		count++;
	}
	return exp2(count);
}

bool isPowerOf2(int input) {
	return (input & (input - 1)) == 0;
}

// Lab4: Device Functions



// Lab4: Kernel Functions
__global__ void singleKernel(float *outArray, float *inArray, int numElements) {
	__shared__ float sharedArray[BLOCK_SIZE];
	sharedArray[threadIdx.x] = (threadIdx.x < numElements) ? inArray[threadIdx.x] : 0;
	__syncthreads();

	for(size_t i = 1; i < (BLOCK_SIZE >> 1); i <<= 1) {
		size_t index = 2 * i * ( threadIdx.x  + 1) - 1;

		if(index < BLOCK_SIZE) {
			sharedArray[index] += sharedArray[index - i];
		}
		__syncthreads();
	}

	if(threadIdx.x == 0) {sharedArray[BLOCK_SIZE - 1] = 0;}
	__syncthreads();

	for(size_t i = BLOCK_SIZE >> 1; i > 0; i >>= 1) {
		size_t index = 2 * i * ( threadIdx.x  + 1) - 1;
		//printf("%d %lu %lu %lu %0.2f %0.2f\n",threadIdx.x, i,   index, index - i, sharedArray[index], sharedArray[index - 1]);

		if(index < BLOCK_SIZE) {
			float temp = sharedArray[index - i];
			sharedArray[index - i] = sharedArray[index];
			sharedArray[index] += temp;
		}
		__syncthreads();
	}

	outArray[threadIdx.x] = sharedArray[threadIdx.x];
}


__global__ void upKernel(float *outArray, float *inArray, int numElements) {
	__shared__ float sharedArray[BLOCK_SIZE];
	sharedArray[threadIdx.x] = inArray[threadIdx.x];
	__syncthreads();

	for(size_t i = 1; i < (numElements >> 1); i <<= 1) {
		size_t index = 2 * i * ( threadIdx.x  + 1) - 1;

		if(index < numElements) {
			sharedArray[index] += sharedArray[index - i];
		}
		__syncthreads();
	}
	if(threadIdx.x == 0) {sharedArray[numElements -1] = 0;}
	__syncthreads();
	outArray[threadIdx.x] = sharedArray[threadIdx.x];
}

__global__ void downKernel(float *outArray, int numElements) {

	__shared__ float sharedArray[BLOCK_SIZE];
	sharedArray[threadIdx.x] = outArray[threadIdx.x];

	if(threadIdx.x == 0) {sharedArray[numElements -1] = 0;}
	__syncthreads();

	for(size_t i = numElements >> 1; i > 0; i>>= 1) {
		size_t index = 2 * i * ( threadIdx.x  + 1) - 1;
		//printf("%d %lu %lu %lu %0.2f %0.2f\n",threadIdx.x, i,   index, index - i, sharedArray[index], sharedArray[index - 1]);

		if(index < numElements) {
			float temp = sharedArray[index - i];
			sharedArray[index - i] = sharedArray[index];
			sharedArray[index] += temp;
		}
		__syncthreads();
	}

	outArray[threadIdx.x] = sharedArray[threadIdx.x];
}



// **===-------- Lab4: Modify the body of this function -----------===**
// You may need to make multiple kernel calls, make your own kernel
// function in this file, and then call them from here.
void prescanArray(float *outArray, float *inArray, int numElements)
{

	if(numElements <= BLOCK_SIZE) {
		singleKernel <<<1, BLOCK_SIZE>>> (outArray, inArray, numElements);
	}

	hipDeviceSynchronize();


}
// **===-----------------------------------------------------------===**


#endif // _PRESCAN_CU_
